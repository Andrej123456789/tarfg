#include "hip/hip_runtime.h"
#include ""

#include "headers/disk.h"

void init_disk()
{
	return;
}

void close_disk()
{
	DISK_FOLDERS.clear();
	DISK_FILES.clear();
}

void add_file(std::string name, std::string content)
{
	bool make_file_in_existing_folder = false;
	size_t pos_i, pos_j = 0;

	std::string file;
	std::string folder;

	size_t pos = name.find('/');
	if (pos != std::string::npos)
	{
		/* fix a bug (if path contains tow or more slashes)*/
		folder = name.substr(0, pos);
		file = name.substr(pos + 1, name.size() - 1);
	}

	else
	{
		file = name;
	}
	
	if (folder.size() > 0)
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				for (size_t j = 0; j < DISK_FOLDERS.at(i).files.size(); j++)
				{
					if (DISK_FOLDERS.at(i).files.at(j).name == file)
					{
						std::cout << "File " << file << " already exists!\n";
						return;
					}

					make_file_in_existing_folder = true;
				}
			}

			pos_i = i;
		}

		if (make_file_in_existing_folder)
		{
			VFILE new_file;

			new_file.name = file;
			new_file.content = content;

			DISK_FOLDERS.at(pos_i).files.push_back(new_file);
		}

		else
		{
			VFOLDER new_folder;
			VFILE new_file;

			new_file.name = file;
			new_file.content = content;

			new_folder.name = folder;
			new_folder.files.push_back(new_file);

			DISK_FOLDERS.push_back(new_folder);
		}
	}

	else
	{
		for (size_t i = 0; i < DISK_FILES.size(); i++)
		{
			if (DISK_FILES.at(i).name == file)
			{
				std::cout << "File " << file << " already exists!\n";
				return;
			}
		}

		VFILE new_file;

		new_file.name = file;
		new_file.content = content;

		DISK_FILES.push_back(new_file);
	}
}

void add_file_from_disk(std::string path, short save)
{
	std::fstream file;
	file.open(path, std::ios::in);

	if (!file)
	{
		std::cout << "File " << path << " not found!\n";
	}

	else
	{
		std::string name;
		
		size_t pos = path.find('/');
		if (pos != std::string::npos)
		{
			/* fix a bug (if path contains tow or more slashes)*/
			name = path.substr(pos + 1, path.size() - 1);
		}

		else
		{
			name = path;
		}

		std::string str;
		std::string content;
        while (std::getline(file, str))
        {
			content += str;

			if (save == 0) /* do not save few bytes for new line */
			{
				content += "\n";
			}
        }

		add_file(name, content);
    }

    file.close();
}

void delete_file(std::string name)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		DISK.erase(DISK.begin() + position);
	}

	else
	{
		std::cout << "File " << name << " not found!\n";
	}
}

void edit_file(std::string name, std::string new_content)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		DISK.at(position).content = new_content;
	}	

	else
	{
		std::cout << "File " << name << " not found!\n";
		std::cout << "Creating new file...\n";

		add_file(name, new_content);
	}

	return 0;
}

int search_file(std::string name)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		auto file = DISK.at(position);

		std::cout << file.name << '\n';
		std::cout << '\t' << file.content << '\n';
	}

	else
	{
		std::cout << "File " << name << " not found!\n";
	}

	return 0;
}
