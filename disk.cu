#include "hip/hip_runtime.h"
#include ""

#include "headers/disk.h"

std::tuple<bool, size_t> check_existence(std::string name)
{
	bool state = false;
	size_t position = 0;
	for (size_t i = 0; i < DISK.size(); i++)
	{
		if (DISK.at(i).name == name)
		{
			state = true;
			position = i;

			break;
		}
	}

	return {state, position};
}

int init_disk()
{
	return 0;
}

int close_disk()
{
	DISK.clear();
	return 0;
}

int add_file(std::string name, std::string content)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		std::cout << "File " << name << " already exists!\n";
		return 0;
	}

	VFILE file;

	file.name = name;
	file.content = content;

	file.size = file.content.size();
	
	DISK.push_back(file);
	return 0;
}

int add_file_from_disk(std::string path, short save)
{
	std::fstream file;
	file.open(path, std::ios::in);

	if (!file)
	{
		std::cout << "File " << path << " not found!\n";
	}

	else
	{
		std::string name;
		
		size_t pos = path.find('/');
		if (pos != std::string::npos)
		{
			/* fix a bug (if path contains tow or more slashes)*/
			name = path.substr(pos + 1, path.size() - 1);
		}

		else
		{
			name = path;
		}

		std::string str;
		std::string content;
        while (std::getline(file, str))
        {
			content += str;

			if (save == 0) /* do not save few bytes for new line */
			{
				content += "\n";
			}
        }

		add_file(name, content);
    }

    file.close();
	return 0;
}

int delete_file(std::string name)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		DISK.erase(DISK.begin() + position);
	}

	else
	{
		std::cout << "File " << name << " not found!\n";
	}

	return 0;
}

int edit_file(std::string name, std::string new_content)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		DISK.at(position).content = new_content;
	}	

	else
	{
		std::cout << "File " << name << " not found!\n";
		std::cout << "Creating new file...\n";

		add_file(name, new_content);
	}

	return 0;
}

int search_file(std::string name)
{
	auto [state, position] = check_existence(name);

	if (state)
	{
		auto file = DISK.at(position);

		std::cout << file.name << '\n';
		std::cout << '\t' << file.content << '\n';
	}

	else
	{
		std::cout << "File " << name << " not found!\n";
	}

	return 0;
}
