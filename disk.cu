#include "hip/hip_runtime.h"
#include ""

#include "headers/disk.h"

void init_disk()
{
	return;
}

void close_disk()
{
	DISK_FOLDERS.clear();
	DISK_FILES.clear();

	return;
}

void add_file(std::string name, std::string content)
{
	bool make_file_in_existing_folder = false;
	size_t pos_i = 0;

	std::string file;
	std::string folder;

	size_t pos = name.find('/');
	if (pos != std::string::npos)
	{
		/* fix a bug (if path contains tow or more slashes)*/
		folder = name.substr(0, pos);
		file = name.substr(pos + 1, name.size() - 1);
	}

	else
	{
		file = name;
	}
	
	if (folder.size() > 0 && file.size() > 0) /* folder and file in name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				for (size_t j = 0; j < DISK_FOLDERS.at(i).files.size(); j++)
				{
					if (DISK_FOLDERS.at(i).files.at(j).name == file)
					{
						std::cout << "File " << file << " already exists!\n";
						return;
					}

					make_file_in_existing_folder = true;
				}
			}

			pos_i = i;
		}

		if (make_file_in_existing_folder) /* folder exists, but not a file */
		{
			for (size_t i = 0; i < DISK_FILES.size(); i++)
			{
				if (DISK_FILES.at(i).name == file)
				{
					std::cout << "File " << file << " already exists!\n";
					return;
				}
			}

			VFILE new_file;

			new_file.name = file;
			new_file.content = content;

			DISK_FOLDERS.at(pos_i).files.push_back(new_file);
		}

		else /* neither folder nor file exist */
		{
			VFOLDER new_folder;
			VFILE new_file;

			new_file.name = file;
			new_file.content = content;

			new_folder.name = folder;
			new_folder.files.push_back(new_file);

			DISK_FOLDERS.push_back(new_folder);
		}
	}

	else if (folder.size() > 0 && file.size() == 0) /* folder in name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				std::cout << "Folder " << folder << " already exists!\n";
				return;
			}

			VFOLDER new_folder;

			new_folder.name = folder;
			DISK_FOLDERS.push_back(new_folder);
		}
	}

	else if (folder.size() == 0 && file.size() > 0) /* file in name */
	{
		for (size_t i = 0; i < DISK_FILES.size(); i++)
		{
			if (DISK_FILES.at(i).name == file)
			{
				std::cout << "File " << file << " already exists!\n";
				return;
			}
		}

		VFILE new_file;

		new_file.name = file;
		new_file.content = content;

		DISK_FILES.push_back(new_file);
	}

	else /* nothing */
	{
		std::cout << "Please enter name of new folder/file!\n";
		return;
	}
}

void add_file_from_disk(std::string path, short save)
{
	std::fstream file;
	file.open(path, std::ios::in);

	if (!file)
	{
		std::cout << "File " << path << " not found!\n";
	}

	else
	{
		std::string name;
		
		size_t pos = path.find('/');
		if (pos != std::string::npos)
		{
			/* fix a bug (if path contains tow or more slashes)*/
			name = path.substr(pos + 1, path.size() - 1);
		}

		else
		{
			name = path;
		}

		std::string str;
		std::string content;
        while (std::getline(file, str))
        {
			content += str;

			if (save == 0) /* do not save few bytes for new line */
			{
				content += "\n";
			}
        }

		add_file(name, content);
    }

    file.close();
}

void delete_file(std::string name)
{
	std::string file;
	std::string folder;

	size_t pos = name.find('/');
	if (pos != std::string::npos)
	{
		/* fix a bug (if path contains tow or more slashes)*/
		folder = name.substr(0, pos);
		file = name.substr(pos + 1, name.size() - 1);
	}

	else
	{
		file = name;
	}
	
	if (folder.size() > 0 && file.size() > 0) /* folder and file in name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				for (size_t j = 0; j < DISK_FOLDERS.at(i).files.size(); j++)
				{
					if (DISK_FOLDERS.at(i).files.at(j).name == file)
					{
						DISK_FOLDERS.at(i).files.erase(DISK_FOLDERS.at(i).files.begin() + j);
						return;
					}
				}
			}
		}

		std::cout << "File " << file << " or folder " << folder << " do not exist and therefore cannot be deleted unless you have PhD in quantum information theory!\n";
		return;
	}

	else if (folder.size() > 0 && file.size() == 0) /* folder in the name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				DISK_FOLDERS.erase(DISK_FOLDERS.begin() + i);
				return;
			}
		}

		std::cout << "Folder " << folder << " does not exist and therefore cannot be deleted unless you have PhD in quantum information theory!\n";
		return;
	}

	else if (folder.size() == 0 && file.size() > 0) /* file in the name */
	{
		for (size_t i = 0; i < DISK_FILES.size(); i++)
		{
			if (DISK_FILES.at(i).name == file)
			{
				DISK_FILES.erase(DISK_FILES.begin() + i);
				return;
			}
		}

		std::cout << "File " << file << " does not exist and therefore cannot be deleted unless you have PhD in quantum information theory!\n";
		return;
	}

	else /* nothing */
	{
		std::cout << "Please enter name of new folder/file!\n";
		return;
	}
}

void edit_file(std::string name, std::string new_content)
{
	delete_file(name);
	add_file(name, new_content);
}

void search_file(std::string name)
{
	bool make_file_in_existing_folder = false;
	size_t pos_i = 0;

	std::string file;
	std::string folder;

	size_t pos = name.find('/');
	if (pos != std::string::npos)
	{
		/* fix a bug (if path contains tow or more slashes)*/
		folder = name.substr(0, pos);
		file = name.substr(pos + 1, name.size() - 1);
	}

	else
	{
		file = name;
	}
	
	if (folder.size() > 0 && file.size() > 0) /* folder and file in name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				for (size_t j = 0; j < DISK_FOLDERS.at(i).files.size(); j++)
				{
					if (DISK_FOLDERS.at(i).files.at(j).name == file)
					{
						std::cout << DISK_FOLDERS.at(i).files.at(j).name << "\n";
						std::cout << '\t' << DISK_FOLDERS.at(i).files.at(j).content << "\n";
						return;
					}

					make_file_in_existing_folder = true;
				}
			}

			pos_i = i;
		}

		if (make_file_in_existing_folder) /* folder exists, but not a file */
		{
			std::cout << DISK_FOLDERS.at(pos_i).name << "\n";

			for (auto &i : DISK_FOLDERS.at(pos_i).files)
			{
				std::cout << '\t' << i.name << "\n";
				std::cout << "\t\t" << i.content << "\n";
			}
		}

		else /* neither folder nor file exist */
		{
			std::cout << "File " << file << " or folder " << folder << " not found!\n";
			return;
		}
	}

	else if (folder.size() > 0 && file.size() == 0) /* folder in name */
	{
		for (size_t i = 0; i < DISK_FOLDERS.size(); i++)
		{
			if (DISK_FOLDERS.at(i).name == folder)
			{
				std::cout << DISK_FOLDERS.at(pos_i).name << "\n";

				for (auto &i : DISK_FOLDERS.at(i).files)
				{
					std::cout << '\t' << i.name << "\n";
					std::cout << "\t\t" << i.content << "\n";
				}
			}
		}
	}

	else if (folder.size() == 0 && file.size() > 0) /* file in name */
	{
		for (size_t i = 0; i < DISK_FILES.size(); i++)
		{
			if (DISK_FILES.at(i).name == file)
			{
				std::cout << file << '\n';
				std::cout << '\t' << DISK_FILES.at(i).content << '\n';
			}
		}
	}

	else /* nothing */
	{
		std::cout << " -------------- FOLDERS -------------- \n";
		for (auto &i : DISK_FOLDERS)
		{
			std::cout << i.name << "\n";

			for (auto &x : i.files)
			{
				std::cout << '\t' << x.name << "\n";
				std::cout << "\t\t" << x.content << "\n";
			}
		}

		std::cout << " -------------- FILES -------------- \n";
		for (auto &j : DISK_FILES)
		{
			std::cout << j.name << '\n';
			std::cout << '\t' << j.content << '\n';
		}
	}
}
