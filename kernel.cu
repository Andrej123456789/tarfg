﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <sstream>
#include <string>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "headers/tarfg.h"
#include "headers/disk.h"

void error(int errorc)
{
    switch (errorc)
    {
        case 0:
            fprintf(stderr, "usage : tarfg <command> <value>\n");
            exit(0);
            break;
    }
}

void split_str(std::string const& str, const char delim, std::vector<std::string>& out)
{
    /* create a stream from the string */
    std::stringstream s(str);

    std::string s2;
    while (getline(s, s2, delim))
    {
        out.push_back(s2); /* store the string in s2 */
    }
}

void shell()
{
    while (true)
    {
        std::string command = "";
        std::string input = "";

again:
        std::cout << ">> ";
        getline(std::cin, input);

        std::vector<std::string> arguments = {};
        const char delim = ' ';
        split_str(input, delim, arguments);
        command = const_cast<char*>(strtok(const_cast<char*>(input.c_str()), " "));

        if (strcmp(command.c_str(), "help") == 0)
        {

        }

        else if (strcmp(command.c_str(), "exit") == 0)
        {
            exit(0);
        }

        else if (command == "tar")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }
        }

        else if (command == "untar")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            untar_file(arguments[1].c_str());
        }

        else if (command == "add")
        {
            if (arguments.size() < 3)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            std::string argument;
            for (size_t i = 0; i < arguments.size(); i++)
            {
                if (i < 2)
                {
                    continue;
                }

                else
                {
                    argument += arguments[i];
                    argument += " ";
                }
            }

            add_file(arguments[1], argument);
        }

        else if (command == "add_disk")
        {
            if (arguments.size() < 3)
            {
                std::cout << "not enough arguments!\n";
                goto again;
            }

            add_file_from_disk(arguments[1], std::stoi(arguments[2]));
        }

        else if (command == "delete")
        {
            if (arguments.size() < 3)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            delete_file(arguments[1]);
        }

        else if (command == "edit")
        {
            if (arguments.size() < 3)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            std::string argument;
            for (size_t i = 0; i < arguments.size(); i++)
            {
                if (i < 2)
                {
                    continue;
                }

                else
                {
                    argument += arguments[i];
                    argument += " ";
                }
            }

            edit_file(arguments[1], argument);
        }

        else if (command == "search")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            search_file(arguments[1]);
        }

        else
        {
            std::cout << "Invalid command! See help for avabible commands!\n";
        }

        input = "";
    }
}

int main()
{
    char cwd[256];
    getcwd(cwd, sizeof(cwd));

    std::cout << cwd << std::endl;

    init_disk();
    shell();

    close_disk();
	return 0;
}
