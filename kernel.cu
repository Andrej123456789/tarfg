﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <sstream>
#include <string>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "headers/disk.h"
#include "microtar/microtar.h"

void error(int errorc)
{
    switch (errorc)
    {
        case 0:
            fprintf(stderr, "usage : tarfg <command> <value>\n");
            exit(0);
            break;
    }
}

void split_str(std::string const& str, const char delim, std::vector<std::string>& out)
{
    /* create a stream from the string */
    std::stringstream s(str);

    std::string s2;
    while (getline(s, s2, delim))
    {
        out.push_back(s2); /* store the string in s2 */
    }
}

void tar(const char* name, short save, std::vector<std::string>& FILES)
{
    int error = 0;
    mtar_t tar;

    /* Open archive for writing */
    error = mtar_open(&tar, name, "w");

    if (error < 0)
    {
        printf("%s\n", mtar_strerror(error));
    }

    for (auto &i : FILES)
    {
        std::fstream file;
        file.open(i, std::ios::in);

        if (!file)
        {
            std::cout << "File " << i << " not found!\n";
            break;
        }

        std::string str;
        std::string content;
        while (std::getline(file, str))
        {
            content += str;

            if (save == 0) /* do not save few bytes for new line */
            {
                content += "\n";
            }
        }

        error = mtar_write_file_header(&tar, i.c_str(), content.size());

        if (error < 0)
        {
            printf("%s\n", mtar_strerror(error));
        }

        error = mtar_write_data(&tar, content.c_str(), content.size());

        if (error < 0)
        {
            printf("%s\n", mtar_strerror(error));
        }
    }

    /* Finalize -- this needs to be the last thing done before closing */
    error = mtar_finalize(&tar);

    if (error < 0)
    {
        printf("%s\n", mtar_strerror(error));
    }

    /* Close archive */
    error = mtar_close(&tar);

    if (error < 0)
    {
        printf("%s\n", mtar_strerror(error));
    }
}

void untar(const char* path)
{
    int error = 0;

    mtar_t tar;
    mtar_header_t h;
    char *p;

    std::vector<std::string> FILES;

    /* Open archive for reading */
    error = mtar_open(&tar, path, "r");

    if (error < 0)
    {
        printf("%s\n", mtar_strerror(error));
        return;
    }

    /* Print a string */
    printf("Extracting from %s\n", path);

    /* Print all file names and sizes */
    while ((mtar_read_header(&tar, &h)) != MTAR_ENULLRECORD)
    {
        printf("\t %s (%d bytes)\n", h.name, h.size);
        FILES.push_back(std::string(h.name));

        error = mtar_next(&tar);

        if (error < 0)
        {
            printf("%s\n", mtar_strerror(error));
            return;
        }
    }

    for (auto &i : FILES)
    {
        /* Load and save content of found files */
        error = mtar_find(&tar, i.c_str(), &h);

        if (error < 0)
        {
            printf("%s\n", mtar_strerror(error));
            return;
        }

        p = (char*)calloc(1, h.size + 1);

        if (p == NULL)
        {
            printf("Error while trying to allocate space!\n");
            return;
        }

        error = mtar_read_data(&tar, p, h.size);

        if (error < 0)
        {
            printf("%s\n", mtar_strerror(error));
        }
        
        add_file(i, std::string(p));
    }

    /* Free the string */
    free(p);

    /* Close archive */
    mtar_close(&tar);
}

void shell()
{
    while (true)
    {
        std::string command = "";
        std::string input = "";

again:
        std::cout << ">> ";
        getline(std::cin, input);

        std::vector<std::string> arguments = {};
        const char delim = ' ';
        split_str(input, delim, arguments);
        command = const_cast<char*>(strtok(const_cast<char*>(input.c_str()), " "));

        if (strcmp(command.c_str(), "help") == 0)
        {

        }

        else if (strcmp(command.c_str(), "exit") == 0)
        {
            exit(0);
        }

        else if (command == "tar")
        {
            if (arguments.size() < 4)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            std::vector<std::string> FILES;
            for (size_t i = 0; i < arguments.size(); i++)
            {
                if (i < 3)
                {
                    continue;
                }

                else
                {
                    FILES.push_back(arguments[i]);
                }
            }

            tar(arguments[1].c_str(), std::stoi(arguments[2]), FILES);
        }

        else if (command == "untar")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            untar(arguments[1].c_str());
        }

        else if (command == "add")
        {
            if (arguments.size() < 3)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            std::string argument;
            for (size_t i = 0; i < arguments.size(); i++)
            {
                if (i < 2)
                {
                    continue;
                }

                else
                {
                    argument += arguments[i];
                    argument += " ";
                }
            }

            add_file(arguments[1], argument);
        }

        else if (command == "add_disk")
        {
            if (arguments.size() < 3)
            {
                std::cout << "not enough arguments!\n";
                goto again;
            }

            add_file_from_disk(arguments[1], std::stoi(arguments[2]));
        }

        else if (command == "delete")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            delete_file(arguments[1]);
        }

        else if (command == "edit")
        {
            if (arguments.size() < 3)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            std::string argument;
            for (size_t i = 0; i < arguments.size(); i++)
            {
                if (i < 2)
                {
                    continue;
                }

                else
                {
                    argument += arguments[i];
                    argument += " ";
                }
            }

            edit_file(arguments[1], argument);
        }

        else if (command == "search")
        {
            if (arguments.size() < 2)
            {
                std::cout << "Not enough arguments!\n";
                goto again;
            }

            search_file(arguments[1]);
        }

        else
        {
            std::cout << "Invalid command! See help for avabible commands!\n";
        }

        input = "";
    }
}

int main()
{
    char cwd[256];
    getcwd(cwd, sizeof(cwd));

    std::cout << cwd << std::endl;

    init_disk();
    shell();

    close_disk();
	return 0;
}
